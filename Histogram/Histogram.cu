#include "hip/hip_runtime.h"
#include <wb.h>
// System Includes
#include <stdio.h>
// For cuda runtime apis
#include <hip/hip_runtime.h>
// Math apis
#include <math.h>

// Number of Bins - Fixed for this assignment
#define D_NUM_BINS 	4096
#define D_BLOCK_WIDTH	256

// Defines for the two kernel implementations
#define D_KERNEL_WITH_BIN_SAT	1
#define D_KERNEL_WO_BIN_SAT	2
#define D_KERNEL_TYPE		(D_KERNEL_WITH_BIN_SAT)

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

#if(D_KERNEL_TYPE == D_KERNEL_WO_BIN_SAT)
// Histogram Kernel with Privatization Method and No Saturation
__global__
void computeHistogram_privatized_nosat(unsigned int * dInput,
  unsigned int * dBins, int dInLen)
{
  /* Private copy of the Bins */
  __shared__ unsigned int sm_Bins[D_NUM_BINS];
  /* Bin index */
  unsigned int binIdx;
  /* Thread index */
  int tIdx = threadIdx.x;
  /* Input index */
  int inIdx = threadIdx.x + blockIdx.x * blockDim.x;
  /* Stride */
  int stride = blockDim.x * gridDim.x;
  /* Initialize the Private Bins Copy */
  for(int i=tIdx; i < D_NUM_BINS; i+=blockDim.x)
  {
    sm_Bins[i] = 0;
  }
  /* synchronization */
  __syncthreads();

  /* Compute Histogram */
  while(inIdx<dInLen)
  {
    binIdx = dInput[inIdx];
    if(binIdx < D_NUM_BINS)
    {
      atomicAdd(&(sm_Bins[binIdx]), 1);
    }
    inIdx+=stride;
  }
  /* synchronization */
  __syncthreads();

  /* Update to Global copy */
  for(int j=tIdx; j < D_NUM_BINS; j+=blockDim.x)
  {
    atomicAdd(&(dBins[j]), sm_Bins[j]);
  }
}

// Kernel to Saturate the Bin Values
__global__
void saturateHistoBins(unsigned int * dBins, int dBinLen)
{
  int Idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(Idx < dBinLen)
  {
    dBins[Idx] = min(dBins[Idx], 127);
  }
}
#else
// Histogram Kernel with Privatization Method and No Saturation
__global__
void computeHistogram_privatized_withsat(unsigned int * dInput,
  unsigned int * dBins, int dInLen)
{
  /* Private copy of the Bins */
  __shared__ unsigned int sm_Bins[D_NUM_BINS];
  /* Bin index */
  unsigned int binIdx;
  /* Thread index */
  int tIdx = threadIdx.x;
  /* Input index */
  int inIdx = threadIdx.x + blockIdx.x * blockDim.x;
  /* Stride */
  int stride = blockDim.x * gridDim.x;
  /* Initialize the Private Bins Copy */
  for(int i=tIdx; i < D_NUM_BINS; i+=blockDim.x)
  {
    sm_Bins[i] = 0;
  }
  /* synchronization */
  __syncthreads();

  /* Compute Histogram */
  while(inIdx<dInLen)
  {
    binIdx = dInput[inIdx];
    if(binIdx < D_NUM_BINS)
    {
      atomicAdd(&(sm_Bins[binIdx]), 1);
    }
    inIdx+=stride;
  }
  /* synchronization */
  __syncthreads();

  /* Update to Global copy */
  for(int j=tIdx; j < D_NUM_BINS; j+=blockDim.x)
  {
    atomicAdd(&(dBins[j]), sm_Bins[j]);
  }

  /* Saturate the results to maximum 127 */
  // Though the following saturation will be performed by
  // every thread block, it won't affect the overall
  // Bins result.
  __syncthreads();

  // Use atomicMin for updating the saturated value in 
  // GlobalMem directly
  for(int k=tIdx; k < D_NUM_BINS; k+=blockDim.x)
  {
    atomicMin(&(dBins[k]), 127);
  } 
}
#endif

int main(int argc, char *argv[]) {
  wbArg_t args;
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;
  int inputLenInBytes; // Input length in bytes
  int binsLenInBytes;  // Bins length in bytes
  hipError_t cudaApiErrVal; // CUDA Error Check
  hipError_t cudaKernelErrVal; // CUDA Error Check

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
                                       &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(D_NUM_BINS * sizeof(unsigned int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", D_NUM_BINS);

  //Initialize HostBins array
  for(int i=0; i<D_NUM_BINS; i++)
  {
    hostBins[i] = 0;
  }
  //Calculate input lentgh in bytes for allocating memory on device
  inputLenInBytes = inputLength * sizeof(unsigned int);
  binsLenInBytes = D_NUM_BINS * sizeof(unsigned int);

  wbTime_start(GPU, "Allocating GPU memory.");
  // Allocate GPU memory - START
  cudaApiErrVal = hipMalloc((void **)&deviceInput,inputLenInBytes);

  if(hipSuccess != cudaApiErrVal)
  {
    printf("hipMalloc deviceInput returned error %s (code %d), line(%d)\n",
    hipGetErrorString(cudaApiErrVal), cudaApiErrVal, __LINE__);
    exit(EXIT_FAILURE);
  }

  cudaApiErrVal = hipMalloc((void **)&deviceBins,binsLenInBytes);

  if(hipSuccess != cudaApiErrVal)
  {
    printf("hipMalloc deviceBins returned error %s (code %d), line(%d)\n",
    hipGetErrorString(cudaApiErrVal), cudaApiErrVal, __LINE__);
    exit(EXIT_FAILURE);
  }
  // Allocate GPU memory - END
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  // Copy memory to the GPU - START

  cudaApiErrVal = hipMemcpy(deviceInput, hostInput, inputLenInBytes,
                             hipMemcpyHostToDevice);

  if(hipSuccess != cudaApiErrVal)
  {
    printf("hipMemcpy deviceInput returned error %s (code %d), line(%d)\n",
    hipGetErrorString(cudaApiErrVal), cudaApiErrVal, __LINE__);
    exit(EXIT_FAILURE);
  }

  cudaApiErrVal = hipMemcpy(deviceBins, hostBins, binsLenInBytes,
                             hipMemcpyHostToDevice);
  if(hipSuccess != cudaApiErrVal)
  {
    printf("hipMemcpy deviceBins returned error %s (code %d), line(%d)\n",
    hipGetErrorString(cudaApiErrVal), cudaApiErrVal, __LINE__);
    exit(EXIT_FAILURE);
  }
  // Copy memory to the GPU - END
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // Initialize the grid and block dimensions
  dim3 dimBlock(D_BLOCK_WIDTH);
  dim3 dimGrid((ceil(D_NUM_BINS/float(dimBlock.x))));

  // Launch kernel
  // ----------------------------------------------------------
  wbLog(TRACE, "Launching kernel");

  #if(D_KERNEL_TYPE == D_KERNEL_WO_BIN_SAT)
	  wbTime_start(Compute, "Performing CUDA computation");
	  // Perform kernel computation here
	  computeHistogram_privatized_nosat<<<dimGrid, dimBlock>>>(deviceInput,
	    deviceBins, inputLength);

	  cudaKernelErrVal = hipGetLastError();
	  if(hipSuccess != cudaKernelErrVal)
	  {
	    printf("Failed to launch the cuda kernel %s (code %d), line(%d)\n",
	    hipGetErrorString(cudaKernelErrVal), cudaKernelErrVal, __LINE__);
	    exit(EXIT_FAILURE);
	  }
	  CUDA_CHECK(hipDeviceSynchronize());

	  wbTime_stop(Compute, "Performing CUDA computation");

	  //  Call Saturation kernel
	  // Block and Grid dimension is same as before

	  wbTime_start(Compute, "Performing computation for Saturation");
	  // Perform kernel computation here
	  saturateHistoBins<<<dimGrid, dimBlock>>>(deviceBins, D_NUM_BINS);

	  cudaKernelErrVal = hipGetLastError();
	  if(hipSuccess != cudaKernelErrVal)
	  {
	    printf("Failed to launch the cuda kernel %s (code %d), line(%d)\n",
	    hipGetErrorString(cudaKernelErrVal), cudaKernelErrVal, __LINE__);
	    exit(EXIT_FAILURE);
	  }
	  CUDA_CHECK(hipDeviceSynchronize());
  #else
          wbTime_start(Compute, "Performing CUDA computation");
	  // Perform kernel computation here
	  computeHistogram_privatized_withsat<<<dimGrid, dimBlock>>>(deviceInput,
	    deviceBins, inputLength);

	  cudaKernelErrVal = hipGetLastError();
	  if(hipSuccess != cudaKernelErrVal)
	  {
	    printf("Failed to launch the cuda kernel %s (code %d), line(%d)\n",
	    hipGetErrorString(cudaKernelErrVal), cudaKernelErrVal, __LINE__);
	    exit(EXIT_FAILURE);
	  }
	  CUDA_CHECK(hipDeviceSynchronize());

	  wbTime_stop(Compute, "Performing CUDA computation");
  #endif

  wbTime_stop(Compute, "Performing computation for Saturation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //Copy the GPU memory back to the CPU here
  cudaApiErrVal = hipMemcpy(hostBins, deviceBins, binsLenInBytes,
                             hipMemcpyDeviceToHost);

  if (hipSuccess != cudaApiErrVal)
  {
	  printf("hipMemcpy hostBins returned error %s (code %d), line(%d)\n",
		  hipGetErrorString(cudaApiErrVal), cudaApiErrVal, __LINE__);
	  exit(EXIT_FAILURE);
  }

  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);
  wbTime_stop(GPU, "Freeing GPU Memory");

  // Verify correctness
  // -----------------------------------------------------
  wbSolution(args, hostBins, D_NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}
