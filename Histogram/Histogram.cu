#include "hip/hip_runtime.h"
#include <wb.h>
// System Includes
#include <stdio.h>
// For cuda runtime apis
#include <hip/hip_runtime.h>

// Number of Bins - Fixed for this assignment
#define D_NUM_BINS 4096

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

// Histogram Kernel
__global__
void computeHistogram(const unsigned int * dInput, unsigned int dBins,
  int dInLen)
{
  /* code */
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;
  int dInLenInBytes; // Input length in bytes
  hipError_t cudaApiErrVal; // CUDA Error Check
  hipError_t cudaKernelErrVal; // CUDA Error Check

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
                                       &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  //Calculate input lentgh in bytes for allocating memory on device
  dInLenInBytes = inputLength * sizeof(int);

  wbTime_start(GPU, "Allocating GPU memory.");
  // Allocate GPU memory - START
  cudaApiErrVal = hipMalloc(&deviceInput,dInLenInBytes);

  if(hipSuccess != cudaApiErrVal)
  {
    printf("hipMalloc deviceInput returned error %s (code %d), line(%d)\n",
    hipGetErrorString(cudaApiErrVal), cudaApiErrVal, __LINE__);
    exit(EXIT_FAILURE);
  }

  cudaApiErrVal = hipMalloc(&deviceBins,D_NUM_BINS);

  if(hipSuccess != cudaApiErrVal)
  {
    printf("hipMalloc deviceBins returned error %s (code %d), line(%d)\n",
    hipGetErrorString(cudaApiErrVal), cudaApiErrVal, __LINE__);
    exit(EXIT_FAILURE);
  }
  // Allocate GPU memory - END
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  // Copy memory to the GPU - START

  cudaApiErrVal = hipMemcpy(deviceInput, hostInput, dInLenInBytes,
                             hipMemcpyHostToDevice);

  if(hipSuccess != cudaApiErrVal)
  {
    printf("hipMemcpy deviceInput returned error %s (code %d), line(%d)\n",
    hipGetErrorString(cudaApiErrVal), cudaApiErrVal, __LINE__);
    exit(EXIT_FAILURE);
  }
  // Copy memory to the GPU - END
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // Initialize the grid and block dimensions
  dim3 dimBlock(BLOCK_WIDTH);
  dim3 dimGrid((inputLength/dimBlock.x)+1);

  // Launch kernel
  // ----------------------------------------------------------
  wbLog(TRACE, "Launching kernel");
  wbTime_start(Compute, "Performing CUDA computation");
  // Perform kernel computation here
  computeHistogram<<<dimGrid, dimBlock>>>(deviceInput, deviceBins,
    inputLength);

  cudaKernelErrVal = hipGetLastError();
  if(hipSuccess != cudaKernelErrVal)
  {
    printf("Failed to launch the cuda kernel %s (code %d), line(%d)\n",
    hipGetErrorString(cudaKernelErrVal), cudaKernelErrVal, __LINE__);
    exit(EXIT_FAILURE);
  }
  CUDA_CHECK(hipDeviceSynchronize());

  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //Copy the GPU memory back to the CPU here
  cudaApiErrVal = hipMemcpy(hostBins, deviceBins, D_NUM_BINS,
                             hipMemcpyDeviceToHost);

  if (hipSuccess != cudaApiErrVal)
  {
	  printf("hipMemcpy hostBins returned error %s (code %d), line(%d)\n",
		  hipGetErrorString(cudaApiErrVal), cudaApiErrVal, __LINE__);
	  exit(EXIT_FAILURE);
  }

  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);
  wbTime_stop(GPU, "Freeing GPU Memory");

  // Verify correctness
  // -----------------------------------------------------
  wbSolution(args, hostBins, D_NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}
